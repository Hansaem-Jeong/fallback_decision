//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: BEV_image_terminate.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 31-Aug-2021 16:30:47
//

// Include Files
#include "BEV_image_terminate.h"
#include "BEV_image_data.h"
#include "rt_nonfinite.h"
#include "MWCUSOLVERUtils.hpp"
#include <cstdio>

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void BEV_image_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
  isInitialized_BEV_image = false;
}

//
// File trailer for BEV_image_terminate.cu
//
// [EOF]
//
