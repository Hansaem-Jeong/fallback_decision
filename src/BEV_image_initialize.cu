//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: BEV_image_initialize.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 31-Aug-2021 16:30:47
//

// Include Files
#include "BEV_image_initialize.h"
#include "BEV_image.h"
#include "BEV_image_data.h"
#include "rt_nonfinite.h"
#include "MWCUSOLVERUtils.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void BEV_image_initialize()
{
  just_one_check_not_empty_init();
  out_Prob_ctrv_not_empty_init();
  BEV_image_init();
  cusolverEnsureInitialization();
  hipGetLastError();
  isInitialized_BEV_image = true;
}

//
// File trailer for BEV_image_initialize.cu
//
// [EOF]
//
