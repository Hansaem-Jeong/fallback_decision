//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Decision_Predict.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 01-Sep-2021 16:48:58
//

// Include Files
#include "Decision_Predict.h"
#include "Decision_Predict_data.h"
#include "Decision_Predict_initialize.h"
#include "Decision_Predict_internal_types.h"
#include "DeepLearningNetwork.h"
#include "predict.h"

// Variable Definitions
static aesnet0_0 aesDecNet;

static bool aesDecNet_not_empty;

// Function Definitions
//
// Arguments    : const unsigned char in[367464]
//                float out[7]
// Return Type  : void
//
void Decision_Predict(const unsigned char in[367464], float out[7])
{
  if (!isInitialized_Decision_Predict) {
    Decision_Predict_initialize();
  }
  if (!aesDecNet_not_empty) {
    coder::DeepLearningNetwork_setup(&aesDecNet);
    aesDecNet_not_empty = true;
  }
  coder::DeepLearningNetwork_predict(&aesDecNet, in, out);
}

//
// Arguments    : void
// Return Type  : void
//
void Decision_Predict_init()
{
  aesDecNet_not_empty = false;
}

//
// File trailer for Decision_Predict.cu
//
// [EOF]
//
