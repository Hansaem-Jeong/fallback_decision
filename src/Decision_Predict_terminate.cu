//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Decision_Predict_terminate.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 01-Sep-2021 16:48:58
//

// Include Files
#include "Decision_Predict_terminate.h"
#include "Decision_Predict_data.h"
#include <cstdio>

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void Decision_Predict_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  isInitialized_Decision_Predict = false;
}

//
// File trailer for Decision_Predict_terminate.cu
//
// [EOF]
//
