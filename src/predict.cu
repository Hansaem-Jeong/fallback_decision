#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: predict.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 01-Sep-2021 16:48:58
//

// Include Files
#include "predict.h"
#include "Decision_Predict_internal_types.h"
#include "DeepLearningNetwork.h"
#include "MWCudaDimUtility.hpp"

// Type Definitions
struct cell_wrap_10 {
  float f1[7];
};

struct cell_wrap_6 {
  float f1[367464];
};

struct cell_wrap_9 {
  float f1[367464];
};

// Function Declarations
static __global__ void
DeepLearningNetwork_predict_kernel1(const unsigned char varargin_1[367464],
                                    cell_wrap_6 dataInputsSingle[1]);

static __global__ void
DeepLearningNetwork_predict_kernel2(const cell_wrap_6 dataInputsSingle[1],
                                    cell_wrap_9 inMiniBatchGroup[1]);

static __global__ void
DeepLearningNetwork_predict_kernel3(const cell_wrap_10 outMiniBatchGroup[1],
                                    float varargout_1[7]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char varargin_1[367464]
//                cell_wrap_6 dataInputsSingle[1]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel1(
        const unsigned char varargin_1[367464], cell_wrap_6 dataInputsSingle[1])
{
  unsigned long threadId;
  int i;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 367464) {
    dataInputsSingle[0].f1[i] = static_cast<float>(varargin_1[i]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const cell_wrap_6 dataInputsSingle[1]
//                cell_wrap_9 inMiniBatchGroup[1]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel2(
        const cell_wrap_6 dataInputsSingle[1], cell_wrap_9 inMiniBatchGroup[1])
{
  unsigned long threadId;
  int i;
  int i1;
  int p;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 61UL);
  threadId = (threadId - static_cast<unsigned long>(i)) / 61UL;
  i1 = static_cast<int>(threadId % 251UL);
  threadId = (threadId - static_cast<unsigned long>(i1)) / 251UL;
  p = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(p < 24)) &&
                        (static_cast<int>(i1 < 251)))) &&
      (static_cast<int>(i < 61))) {
    inMiniBatchGroup[0].f1[(i + 61 * i1) + 15311 * p] =
        dataInputsSingle[0].f1[(i1 + 251 * i) + 15311 * p];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const cell_wrap_10 outMiniBatchGroup[1]
//                float varargout_1[7]
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void DeepLearningNetwork_predict_kernel3(
        const cell_wrap_10 outMiniBatchGroup[1], float varargout_1[7])
{
  unsigned long threadId;
  int i;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 7) {
    varargout_1[i] = outMiniBatchGroup[0].f1[i];
  }
}

//
// Arguments    : aesnet0_0 *obj
//                const unsigned char varargin_1[367464]
//                float varargout_1[7]
// Return Type  : void
//
namespace coder {
void DeepLearningNetwork_predict(aesnet0_0 *obj,
                                 const unsigned char varargin_1[367464],
                                 float varargout_1[7])
{
  cell_wrap_10(*gpu_outMiniBatchGroup)[1];
  cell_wrap_6(*gpu_dataInputsSingle)[1];
  cell_wrap_9(*gpu_inMiniBatchGroup)[1];
  float(*gpu_varargout_1)[7];
  unsigned char(*gpu_varargin_1)[367464];
  hipMalloc(&gpu_varargout_1, 28UL);
  hipMalloc(&gpu_outMiniBatchGroup, 28UL);
  hipMalloc(&gpu_inMiniBatchGroup, 1469856UL);
  hipMalloc(&gpu_dataInputsSingle, 1469856UL);
  hipMalloc(&gpu_varargin_1, 367464UL);
  hipMemcpy(*gpu_varargin_1, varargin_1, 367464UL, hipMemcpyHostToDevice);
  DeepLearningNetwork_predict_kernel1<<<dim3(718U, 1U, 1U),
                                        dim3(512U, 1U, 1U)>>>(
      *gpu_varargin_1, *gpu_dataInputsSingle);
  DeepLearningNetwork_predict_kernel2<<<dim3(718U, 1U, 1U),
                                        dim3(512U, 1U, 1U)>>>(
      *gpu_dataInputsSingle, *gpu_inMiniBatchGroup);
  hipMemcpy(obj->getInputDataPointer(0), (*gpu_inMiniBatchGroup)[0].f1,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy((*gpu_outMiniBatchGroup)[0].f1, obj->getLayerOutput(9, 0),
             obj->getLayerOutputSize(9, 0), hipMemcpyDeviceToDevice);
  DeepLearningNetwork_predict_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_outMiniBatchGroup, *gpu_varargout_1);
  hipMemcpy(varargout_1, *gpu_varargout_1, 28UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_varargin_1);
  hipFree(*gpu_dataInputsSingle);
  hipFree(*gpu_inMiniBatchGroup);
  hipFree(*gpu_outMiniBatchGroup);
  hipFree(*gpu_varargout_1);
}

} // namespace coder

//
// File trailer for predict.cu
//
// [EOF]
//
