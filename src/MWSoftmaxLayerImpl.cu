#include "MWSoftmaxLayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWTargetNetworkImpl.hpp"
 MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&shEncNmxJsMuJKwbrwok)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&sjLjZacPSDNBEjAccrGU)); } 
MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() { } void 
MWSoftmaxLayerImpl::propagateSize() { MWCNNLayer* sfmxLayer = getLayer(); 
MWTensorBase* ipTensor = sfmxLayer->getInputTensor(0); MWTensorBase* opTensor = 
sfmxLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(shEncNmxJsMuJKwbrwok, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, ipTensor->getSequenceLength()*ipTensor->getBatchSize(), 
ipTensor->getChannels(), ipTensor->getHeight(), ipTensor->getWidth())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(sjLjZacPSDNBEjAccrGU, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, opTensor->getSequenceLength()*opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
MWSoftmaxLayerImpl::predict() { MWCNNLayer* sfmxLayer = getLayer(); 
MWTensorBase* ipTensorBase = sfmxLayer->getInputTensor(0); MWTensorBase* 
opTensorBase = sfmxLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnSoftmaxForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), 
shEncNmxJsMuJKwbrwok, ipTensor->getData(), getZeroPtr(), 
sjLjZacPSDNBEjAccrGU, opTensor->getData())); } void 
MWSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(shEncNmxJsMuJKwbrwok)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(sjLjZacPSDNBEjAccrGU)); }