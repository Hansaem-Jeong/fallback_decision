#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 01-Sep-2021 16:48:58
//

// Include Files
#include "DeepLearningNetwork.h"
#include "Decision_Predict_internal_types.h"
#include "MWCNNLayer.hpp"
#include "MWElementwiseAffineLayer.hpp"
#include "MWFCLayer.hpp"
#include "MWFusedConvReLULayer.hpp"
#include "MWInputLayer.hpp"
#include "MWMaxPoolingLayer.hpp"
#include "MWOutputLayer.hpp"
#include "MWSoftmaxLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "MWTensor.hpp"
#include "MWTensorBase.hpp"
#include <cstdio>


// Named Constants
const char *errorString{
    "Abnormal termination due to: %s.\nError in %s (line %d)."};

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line);

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::allocate()
{
  this->targetImpl->allocate(367464, 2);
  for (int idx{0}; idx < 10; idx++) {
    this->layers[idx]->allocate();
  }
  (static_cast<MWTensor<float> *>(this->inputTensors[0]))
      ->setData(this->layers[0]->getLayerOutput(0));
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::cleanup()
{
  this->deallocate();
  for (int idx{0}; idx < 10; idx++) {
    this->layers[idx]->cleanup();
  }
  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::deallocate()
{
  this->targetImpl->deallocate();
  for (int idx{0}; idx < 10; idx++) {
    this->layers[idx]->deallocate();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::postsetup()
{
  this->targetImpl->postSetup(this->layers, this->numLayers);
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::resetState()
{
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::setSize()
{
  for (int idx{0}; idx < 10; idx++) {
    this->layers[idx]->propagateSize();
  }
  this->allocate();
  this->postsetup();
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::setup()
{
  if (this->isInitialized) {
    this->resetState();
  } else {
    this->isInitialized = true;
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))
        ->createInputLayer(this->targetImpl, this->inputTensors[0], 251, 61, 24,
                           0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
        ->createElementwiseAffineLayer(
            this->targetImpl, this->layers[0]->getOutputTensor(0), 1, 1, 24, 1,
            1, 24, false, 1, 1,
            "./src/fallback_decision/src/cnn_aesnet0_0_imageinput_scale.bin",
            "./src/fallback_decision/src/"
            "cnn_aesnet0_0_imageinput_offset.bin",
            0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[1]->getOutputTensor(0), 3, 3,
                                   24, 8, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./src/fallback_decision/src/"
                                   "cnn_aesnet0_0_Convolution Layer 1_w.bin",
                                   "./src/fallback_decision/src/"
                                   "cnn_aesnet0_0_Convolution Layer 1_b.bin",
                                   1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[3]))
        ->createMaxPoolingLayer<float, float>(
            this->targetImpl, this->layers[2]->getOutputTensor(0), 2, 2, 2, 2,
            0, 0, 0, 0, 0, 0, "FLOAT", 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[4]))
        ->createFusedConvReLULayer(
            this->targetImpl, 1, this->layers[3]->getOutputTensor(0), 3, 3, 8,
            16, 1, 1, 1, 1, 1, 1, 1, 1, 1,
            "./src/fallback_decision/src/cnn_aesnet0_0_conv_1_w.bin",
            "./src/fallback_decision/src/cnn_aesnet0_0_conv_1_b.bin", 1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[5]))
        ->createMaxPoolingLayer<float, float>(
            this->targetImpl, this->layers[4]->getOutputTensor(0), 2, 2, 2, 2,
            0, 0, 0, 0, 0, 0, "FLOAT", 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[6]))
        ->createFusedConvReLULayer(
            this->targetImpl, 1, this->layers[5]->getOutputTensor(0), 3, 3, 16,
            32, 1, 1, 1, 1, 1, 1, 1, 1, 1,
            "./src/fallback_decision/src/cnn_aesnet0_0_conv_2_w.bin",
            "./src/fallback_decision/src/cnn_aesnet0_0_conv_2_b.bin", 1);
    (static_cast<MWFCLayer *>(this->layers[7]))
        ->createFCLayer(this->targetImpl, this->layers[6]->getOutputTensor(0),
                        29760, 7,
                        "./src/fallback_decision/src/cnn_aesnet0_0_fully "
                        "Connected Layer_w.bin",
                        "./src/fallback_decision/src/cnn_aesnet0_0_fully "
                        "Connected Layer_b.bin",
                        0);
    (static_cast<MWSoftmaxLayer *>(this->layers[8]))
        ->createSoftmaxLayer(this->targetImpl,
                             this->layers[7]->getOutputTensor(0), 1);
    (static_cast<MWOutputLayer *>(this->layers[9]))
        ->createOutputLayer(this->targetImpl,
                            this->layers[8]->getOutputTensor(0), 1);
    this->outputTensors[0] = this->layers[9]->getOutputTensor(0);
    this->setSize();
  }
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line)
{
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    printf(errorString, hipGetErrorString(errCode), file, line);
  }
}

//
// Arguments    : void
// Return Type  : ::aesnet0_0
//
aesnet0_0::aesnet0_0()
{
  this->numLayers = 10;
  this->isInitialized = false;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("imageinput");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("imageinput_normalization");
  this->layers[1]->setInPlaceIndex(0, 0);
  this->layers[2] = new MWFusedConvReLULayer;
  this->layers[2]->setName("Convolution Layer 1_relu Layer 1");
  this->layers[3] = new MWMaxPoolingLayer;
  this->layers[3]->setName("max Pooling Layer 1");
  this->layers[4] = new MWFusedConvReLULayer;
  this->layers[4]->setName("conv_1_relu Layer 2");
  this->layers[5] = new MWMaxPoolingLayer;
  this->layers[5]->setName("max Pooling Layer 2");
  this->layers[6] = new MWFusedConvReLULayer;
  this->layers[6]->setName("conv_2_relu Layer 3");
  this->layers[7] = new MWFCLayer;
  this->layers[7]->setName("fully Connected Layer");
  this->layers[8] = new MWSoftmaxLayer;
  this->layers[8]->setName("softmax Layer");
  this->layers[9] = new MWOutputLayer;
  this->layers[9]->setName("classoutput");
  this->layers[9]->setInPlaceIndex(0, 0);
  this->targetImpl = new MWTargetNetworkImpl;
  this->inputTensors[0] = new MWTensor<float>;
  this->inputTensors[0]->setHeight(251);
  this->inputTensors[0]->setWidth(61);
  this->inputTensors[0]->setChannels(24);
  this->inputTensors[0]->setBatchSize(1);
  this->inputTensors[0]->setSequenceLength(1);
}

//
// Arguments    : void
// Return Type  : void
//
aesnet0_0::~aesnet0_0()
{
  this->cleanup();
  checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (int idx{0}; idx < 10; idx++) {
    delete this->layers[idx];
  }
  if (this->targetImpl) {
    delete this->targetImpl;
  }
  delete this->inputTensors[0];
}

//
// Arguments    : void
// Return Type  : int
//
int aesnet0_0::getBatchSize()
{
  return this->inputTensors[0]->getBatchSize();
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *aesnet0_0::getInputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(this->inputTensors[b_index]))
      ->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *aesnet0_0::getInputDataPointer()
{
  return (static_cast<MWTensor<float> *>(this->inputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *aesnet0_0::getLayerOutput(int layerIndex, int portIndex)
{
  return this->layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int aesnet0_0::getLayerOutputSize(int layerIndex, int portIndex)
{
  return this->layers[layerIndex]
             ->getOutputTensor(portIndex)
             ->getNumElements() *
         sizeof(float);
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *aesnet0_0::getOutputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(this->outputTensors[b_index]))
      ->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *aesnet0_0::getOutputDataPointer()
{
  return (static_cast<MWTensor<float> *>(this->outputTensors[0]))->getData();
}

//
// Arguments    : void
// Return Type  : void
//
void aesnet0_0::predict()
{
  for (int idx{0}; idx < 10; idx++) {
    this->layers[idx]->predict();
  }
}

//
// Arguments    : aesnet0_0 *obj
// Return Type  : void
//
namespace coder {
void DeepLearningNetwork_setup(aesnet0_0 *obj)
{
  obj->setup();
}

} // namespace coder

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
